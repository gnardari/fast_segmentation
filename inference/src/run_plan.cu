#include "hip/hip_runtime.h"
#include <iostream>
#include <algorithm>
#include <chrono>
#include <cstdlib>
#include <fstream>
#include <iostream>
#include <string>
#include <sys/stat.h>
#include <unordered_map>
#include <cassert>
#include <vector>

// #include <hip/hip_runtime_api.h>
#include "NvInfer.h"
#include "NvUffParser.h"
#include "NvUtils.h"

#include <opencv2/opencv.hpp>

// #include "logger.h"
// #include "common.h"

using namespace nvuffparser;
using namespace nvinfer1;
using namespace std;

class Logger : public ILogger           
 {
     void log(Severity severity, const char* msg) override
     {
         // suppress info-level messages
         if (severity != Severity::kINFO)
             std::cout << msg << std::endl;
     }
 } gLogger;

float *imageToTensor(const cv::Mat & image)
{
  const size_t height = image.rows;
  const size_t width = image.cols;
  const size_t channels = image.channels();
  const size_t numel = height * width * channels;

  const size_t stridesCv[3] = { width * channels, channels, 1 };
  const size_t strides[3] = { height * width, width, 1 };

  float * tensor;
  hipHostAlloc((void**)&tensor, numel * sizeof(float), hipHostMallocMapped);

  for (int i = 0; i < height; i++) 
  {
    for (int j = 0; j < width; j++) 
    {
      for (int k = 0; k < channels; k++) 
      {
        const size_t offsetCv = i * stridesCv[0] + j * stridesCv[1] + k * stridesCv[2];
        const size_t offset = k * strides[0] + i * strides[1] + j * strides[2];
        tensor[offset] = (float) image.data[offsetCv];
      }
    }
  }

  return tensor;
}

/* float* createCudaBuffer(int height, int width, int num_classes, */
/*         string fname, bool map_mem){ */
/*   cv::Mat image = cv::imread(fname, 0); */
/*   cv::resize(image, image, cv::Size(width, height)); */
/*   float *input = imageToTensor(image); */
/*  */
/*   float *output; */
/*   float *inputDevice; */
/*   float *outputDevice; */
/*   size_t inputSize = height * width * image.channels() * sizeof(float); */
/*  */
/*   hipHostAlloc(&output, num_classes * sizeof(float), hipHostMallocMapped); */
/*  */
/*   if (map_mem){ */
/*     hipHostGetDevicePointer(&inputDevice, input, 0); */
/*     hipHostGetDevicePointer(&outputDevice, output, 0); */
/*   } */
/*   else { */
/*     hipMalloc(&inputDevice, inputSize); */
/*     hipMalloc(&outputDevice, num_classes * sizeof(float)); */
/*   } */
/*  */
/*   string input_tensor = "inputs/X"; */
/*   string output_tensor = "up23/BiasAdd"; */
/*   int inputBindIndex = engine->getBindingIndex(input_tensor.c_str()); */
/*   int outputBindIndex = engine->getBindingIndex(output_tensor.c_str()); */
/*   float *bindings[2]; */
/*   bindings[inputBindingIndex] = inputDevice; */
/*   bindings[outputBindingIndex] = outputDevice; */
/*   return &bindings; */
/* } */

size_t argmax(float *tensor, size_t numel)
{
  if (numel <= 0)
    return 0;

  size_t totalA = 0;
  size_t totalB = 0;
  vector<int> max;
  for (int i = 0; i < 16*3600*2; i+=2){
      if (tensor[i] < tensor[i+1]){
          max.push_back(1);
          totalA++;
      } else {
          max.push_back(0);
          totalB++;
      }
  }

  cout << "A: " << totalA << " B: " << totalB << endl;
  return 0;
}

void run(){
    IRuntime* runtime = createInferRuntime(gLogger);
    int num_runs = 1000;
    int width = 3600;
    int height = 16;
    int num_classes = 2;
    bool map_mem = true;
    string planPath = "/home/jetson/Documents/realtime_segmentation/models/simple_erfnet.plan";
    string imagePath = "/home/jetson/Documents/realtime_segmentation/data/input.jpg";
    string input_tensor = "inputs/X";
    string output_tensor = "up23/BiasAdd";

    cout << "Starting Inference" << endl;

    ifstream planFile(planPath);
    stringstream planBuffer;
    planBuffer << planFile.rdbuf();
    string plan = planBuffer.str();

    ICudaEngine *engine = runtime->deserializeCudaEngine((void*)plan.data(),
      plan.size(), nullptr);
    IExecutionContext *context = engine->createExecutionContext();

    cout << "Initialized Engine" << endl;
    /*
    THIS SHOULD BE A FUNCTION
    */

    cv::Mat image = cv::imread(imagePath, 0);
    cv::resize(image, image, cv::Size(width, height));
    float *input = imageToTensor(image);

    cout << "Read Image" << endl;
    float *output;
    float *inputDevice;
    float *outputDevice;
    size_t inputSize = height * width * image.channels() * sizeof(float);

    hipHostAlloc(&output, num_classes * sizeof(float), hipHostMallocMapped);
    cout << "Memory Allocation" << endl;

    if (map_mem){
      hipHostGetDevicePointer(&inputDevice, input, 0);
      hipHostGetDevicePointer(&outputDevice, output, 0);
      cout << "Mapped GPU and host memory" << endl;
    }
    else {
      hipMalloc(&inputDevice, inputSize);
      hipMalloc(&outputDevice, num_classes * sizeof(float));
    }

    int inputBindIndex = engine->getBindingIndex(input_tensor.c_str());
    int outputBindIndex = engine->getBindingIndex(output_tensor.c_str());
    float *bindings[2];
    bindings[inputBindIndex] = inputDevice;
    bindings[outputBindIndex] = outputDevice;
    cout << "Got Bindings" << endl;

    //float *bindings = createCudaBuffer(256, 256, 2, imagePath, true);

    double avgTime = 0;

    for (int i = 0; i < num_runs + 1; i++){
        chrono::duration<double> diff;
        auto t0 = chrono::steady_clock::now();
        context->execute(1, (void**)bindings);
        //cout << "Ran execute" << endl;
        auto t1 = chrono::steady_clock::now();
        diff = t1 - t0;

    if (i != 0)
      avgTime += diff.count()*1000.0;
    }
    avgTime /= num_runs;
    cout << "Average inference time: " << avgTime << "ms" << endl;

    argmax(output, num_classes);

    hipFree(inputDevice);
    hipFree(outputDevice);

    hipHostFree(input);
    hipHostFree(output);

    engine->destroy();
    context->destroy();
    runtime->destroy();
}

int main(int argc, char * argv[]){
  run();
  return 0;
}
