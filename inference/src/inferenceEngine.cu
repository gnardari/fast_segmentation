#include "hip/hip_runtime.h"
#include <inferenceEngine.h>

InferenceEngine::InferenceEngine(EngConfig ec){

    modelPath_ = ec.planPath;
    inputTensorName_ = ec.inputTensorName;
    outputTensorName_ = ec.outputTensorName;
    width_ = ec.width;
    height_ = ec.height;
    numClasses_ = ec.numClasses;
    inputSizeBytes_ = height_ * width_ * sizeof(float);

    ifstream planFile(modelPath_);
    stringstream planBuffer;
    planBuffer << planFile.rdbuf();
    string plan = planBuffer.str();

    Logger gLogger;
    runtime_ = createInferRuntime(gLogger);
    engine_ = runtime_->deserializeCudaEngine((void*)plan.data(),
      plan.size(), nullptr);

    context_ = engine_->createExecutionContext();

    // Assuming output has the same height/width as the same as input
    hipHostAlloc(&output_,
            inputSizeBytes_ * numClasses_, hipHostMallocMapped);
    hipHostGetDevicePointer(&outputDevice_, output_, 0);

    inputBindIndex_ = engine_->getBindingIndex(inputTensorName_.c_str());
    outputBindIndex_ = engine_->getBindingIndex(outputTensorName_.c_str());

    bindings_[outputBindIndex_] = outputDevice_;
}

InferenceEngine::~InferenceEngine(void){
    hipFree(inputDevice_);
    hipFree(outputDevice_);

    hipHostFree(output_);

    engine_->destroy();
    context_->destroy();
    runtime_->destroy();
}
float* InferenceEngine::imageToTensor_(const cv::Mat & image)
{
  const unsigned int channels = image.channels();
  const unsigned int strides[3] = { height_ * width_, width_, 1 };

  float* tensor;
  hipHostAlloc((void**)&tensor, channels*inputSizeBytes_, hipHostMallocMapped);

  for (int i = 0; i < height_; i++) {
    const float* row_ptr = image.ptr<float>(i);
    for (int j = 0; j < width_; j++) {
        const unsigned int offset = i * strides[1] + j * strides[2];
        tensor[offset] = (float) row_ptr[j];
    }
  }
  return tensor;
}
/*
float* InferenceEngine::imageToTensor_(const cv::Mat & image)
{
  const unsigned int channels = image.channels();
  const unsigned int stridesCv[3] = { width_ * channels, channels, 1 };
  const unsigned int strides[3] = { height_ * width_, width_, 1 };

  float * tensor;
  hipHostAlloc((void**)&tensor, channels*inputSizeBytes_, hipHostMallocMapped);

  for (int i = 0; i < height_; i++) {
    for (int j = 0; j < width_; j++) {
      for (int k = 0; k < channels; k++) {
        const unsigned int offsetCv = i * stridesCv[0] + j * stridesCv[1] + k * stridesCv[2];
        const unsigned int offset = k * strides[0] + i * strides[1] + j * strides[2];
        tensor[offset] = (float) image.data[offsetCv];
      }
    }
  }

  return tensor;
}
*/
unsigned int InferenceEngine::countClasses_(float *tensor){
    // this function is just for debugging
    unsigned int totalA = 0;
    unsigned int totalB = 0;
    vector<int> max;
    for (int i = 0; i < 16*3600*2; i+=2){
      if (tensor[i] < tensor[i+1]){
          max.push_back(1);
          totalA++;
      } else {
          max.push_back(0);
          totalB++;
      }
    }

    cout << "A: " << totalA << " B: " << totalB << endl;
    return 0;
}

void InferenceEngine::argmax_(float *tensor, vector<unsigned char>& max){
  unsigned int outSize = height_*width_*numClasses_;
  for (unsigned int i = 0; i < outSize; i += numClasses_){
      unsigned int maxIdx = i;
      unsigned char outIdx = 0;
      for(unsigned int c = 1; c < numClasses_; c++){
        if(tensor[maxIdx] < tensor[i+c]){
            maxIdx = i+c;
            outIdx = c;
        }
      }
      max.push_back(outIdx*255);
  }
}

void InferenceEngine::run(const cv::Mat & image, cv::Mat& out){
    timer_.startCpuTimer();
    float* input = imageToTensor_(image);
    timer_.endCpuTimer();
    std::cout << "input preproc: " << timer_.getCpuElapsedTimeForPreviousOperation() << std::endl;

    timer_.startGpuTimer();
    hipHostGetDevicePointer(&inputDevice_, input, 0);
    bindings_[inputBindIndex_] = inputDevice_;
    timer_.endGpuTimer();
    std::cout << "bindings: " << timer_.getGpuElapsedTimeForPreviousOperation() << std::endl;

    timer_.startGpuTimer();
    context_->execute(1, (void**)bindings_);
    timer_.endGpuTimer();
    std::cout << "execute model: " << timer_.getGpuElapsedTimeForPreviousOperation() << std::endl;
    
    vector<unsigned char> max;
    //countClasses_(output_);
    argmax_(output_, max);

    //memcpy(out.data, input, 16*3600*sizeof(float));
    hipHostFree(input);

    memcpy(out.data, max.data(), max.size()*sizeof(unsigned char));
}
